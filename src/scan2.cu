#include "hip/hip_runtime.h"
#include "scan2.h"
#include "gpuerrors.h"
#include "stdio.h"
#include "math.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z
__global__ void Kernel1(int *input, int x)
{
    int tid = (bz*gridDim.y*gridDim.x + by * gridDim.x + bx) * blockDim.x  + tx;
//int tid = (by * gridDim.x + bx) * blockDim.x  + tx;
	input[tid]=input[tid]+x;



}
__global__ void Kernel(int *input, int *output,int* results, int space, int step, int steps, bool Direction)
{
//int i = (by * gridDim.x + bx) * blockDim.x  + tx;
    int tid = (bz*gridDim.y*gridDim.x + by * gridDim.x + bx) * blockDim.x  + tx;

//int tix = threadIdx.x + blockDim.x * blockIdx.x; 
//int tiy = threadIdx.y + blockDim.y * blockIdx.y;
//int tid = tix + tiy*gridDim.x*blockDim.x;




int res ;
	if(Direction)
	{
		if(tid<space)
		{
			res  = output[tid]; //ONLY REWRITE TO CORRECT MEMORY ADDRESS
			input[tid] = res;
		}
		else
		{
			res = output[tid] + output[tid-space];
			input[tid] = res;
		}
	}
	else
	{
		if(tid<space)
		{
			res = input[tid]; //ONLY REWRITE TO CORRECT MEMORY ADRESS
			output[tid] = res;
		}
		else
		{
			res = input[tid] + input[tid-space];
			output[tid] = res;
		}
	}

if (step == 0) {
results[tid] = -1 * input[tid];
}
if(step == (steps-1)){
results[tid] += res;
}

}


void gpuKernel(int* a, int* c, int n) {	
        int* input1;
	int* output1;
	int* result1;



       
if(n<67108864)
{
	HANDLE_ERROR(hipMalloc((void**)&input1, n * sizeof(int)));
        HANDLE_ERROR(hipMalloc((void**)&output1, n * sizeof(int)));
        HANDLE_ERROR(hipMalloc((void**)&result1, n * sizeof(int)));
        HANDLE_ERROR(hipMemcpy(input1, a, n * sizeof(int), hipMemcpyHostToDevice));

dim3 THREADS_PER_BLOCK(1024,1,1);
	dim3 BLOCKS_PER_GRID( n / 1048576, 32,32);	


//LAUNCH KERNELS IN LOOP
int space = 1;

bool direction=0;
	int steps = (int)log2((float)n);
for ( int step = 0; step<steps; step++){
 direction = ((step % 2) !=0);  
Kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(input1, output1, result1, space,  step, steps,direction);
space = space * 2;
}


	

	HANDLE_ERROR(hipMemcpy(c, result1, n * sizeof(int), hipMemcpyDeviceToHost));

        HANDLE_ERROR(hipFree(input1));
        HANDLE_ERROR(hipFree(output1));
        HANDLE_ERROR(hipFree(result1));
}


if(n==67108864)
{
	HANDLE_ERROR(hipMalloc((void**)&input1, n * sizeof(int)));
        HANDLE_ERROR(hipMalloc((void**)&output1, n * sizeof(int)));
        HANDLE_ERROR(hipMalloc((void**)&result1, n * sizeof(int)));
        HANDLE_ERROR(hipMemcpy(input1, a, n * sizeof(int), hipMemcpyHostToDevice));

dim3 THREADS_PER_BLOCK(1024/8,1,1);
	dim3 BLOCKS_PER_GRID( n / 1048576, 32*2,32*4);	




//LAUNCH KERNELS IN LOOP
int space = 1;

bool direction=0;
	int steps = (int)log2((float)n);
for ( int step = 0; step<steps; step++){
 direction = ((step % 2) !=0);  
Kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(input1, output1, result1, space,  step, steps,direction);
space = space * 2;
}


	

	HANDLE_ERROR(hipMemcpy(c, result1, n * sizeof(int), hipMemcpyDeviceToHost));

        HANDLE_ERROR(hipFree(input1));
        HANDLE_ERROR(hipFree(output1));
        HANDLE_ERROR(hipFree(result1));
}






if(n==67108864*2)
{
	HANDLE_ERROR(hipMalloc((void**)&input1,  (n/2) * sizeof(int)));
        HANDLE_ERROR(hipMalloc((void**)&output1, (n/2) * sizeof(int)));
        HANDLE_ERROR(hipMalloc((void**)&result1, (n/2) * sizeof(int)));
	//int* t1;
	//int* c1;
        //t1 = (int*)malloc((n/2) * sizeof(int));
       // c1 = (int*)malloc((n/2) * sizeof(int));
	

//for(int j =0; j< n/2 ;j++)
//	t1[j]=a[j];



        HANDLE_ERROR(hipMemcpy(input1, a, (n/2) * sizeof(int), hipMemcpyHostToDevice));


	//dim3 THREADS_PER_BLOCK(1024/8,1,1);
	//dim3 BLOCKS_PER_GRID( n / 1048576, 16,16);	
dim3 THREADS_PER_BLOCK(1024/8,1,1);
	dim3 BLOCKS_PER_GRID( n / (1048576*2), 64,64*2);	

int space = 1;

bool direction=0;
	int steps = (int)log2((float)(n/2));
for ( int step = 0; step<steps; step++){
 direction = ((step % 2) !=0);  
Kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(input1, output1, result1, space,  step, steps,direction);
space = space * 2;
}




//printf("11111");
	HANDLE_ERROR(hipMemcpy(c, result1, (n/2) * sizeof(int), hipMemcpyDeviceToHost));

//for(int j =0; j< n/2 ;j++)
//	c[j]=t1[j];

int carry = c[(n/2)-1]+a[(n/2)-1];
//////////////////////////////////////////////////
//for(int j =0; j< n/2 ;j++)
//	t1[j]=a[j+(n/2)];

HANDLE_ERROR(hipMemcpy(input1, &a[n/2], n/2 * sizeof(int), hipMemcpyHostToDevice));


	//dim3 THREADS_PER_BLOC1(1024,1,1);
	//dim3 BLOCKS_PER_GRID1( n / 1048576,32,32);
 space = 1;

 direction=0;
	 steps = (int)log2((float)n/2);
for (int step = 0; step<steps; step++){
 direction = ((step % 2) !=0);  
Kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(input1, output1, result1, space,  step, steps,direction);
space = space * 2;
}
//t1=&result1[n/2];
dim3 b(1024,1,1);
	dim3 a( n / (1024*1024),256,1);
	Kernel1<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(result1,carry);
//Kernel1<<<a,b >>>(result1,carry);
//printf("fghdghdhdhdh");
	HANDLE_ERROR(hipMemcpy(&c[n/2], result1, n/2 * sizeof(int), hipMemcpyDeviceToHost));

//for(int j =0; j< n/2 ;j++)
//	c[j+n/2]+=carry;



        HANDLE_ERROR(hipFree(input1));
        HANDLE_ERROR(hipFree(output1));
        HANDLE_ERROR(hipFree(result1));

}

if(n==67108864*4)
{
	HANDLE_ERROR(hipMalloc((void**)&input1,  (n/4) * sizeof(int)));
        HANDLE_ERROR(hipMalloc((void**)&output1, (n/4) * sizeof(int)));
        HANDLE_ERROR(hipMalloc((void**)&result1, (n/4) * sizeof(int)));
	
        HANDLE_ERROR(hipMemcpy(input1, a, (n/4) * sizeof(int), hipMemcpyHostToDevice));

	
dim3 THREADS_PER_BLOCK(1024/8,1,1);
	dim3 BLOCKS_PER_GRID( n / (1048576*4), 64,64*2);	

int space = 1;

bool direction=0;
	int steps = (int)log2((float)(n/4));
for ( int step = 0; step<steps; step++){
 direction = ((step % 2) !=0);  
Kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(input1, output1, result1, space,  step, steps,direction);
space = space * 2;
}

	HANDLE_ERROR(hipMemcpy(c, result1, (n/4) * sizeof(int), hipMemcpyDeviceToHost));

int carry = c[(n/4)-1]+a[(n/4)-1];
//////////////////////////////////////////////////

HANDLE_ERROR(hipMemcpy(input1, &a[n/4], n/4 * sizeof(int), hipMemcpyHostToDevice));

 space = 1;
 direction=0;
 steps = (int)log2((float)n/4);
for (int step = 0; step<steps; step++){
 direction = ((step % 2) !=0);  
Kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(input1, output1, result1, space,  step, steps,direction);
space = space * 2;
}
	Kernel1<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(result1,carry);

	HANDLE_ERROR(hipMemcpy(&c[n/4], result1, n/4 * sizeof(int), hipMemcpyDeviceToHost));

//for(int j =0; j< n/4 ;j++)
//	c[j+n/4]+=carry;

 carry = c[(n/2)-1]+a[(n/2)-1];
//////////////////////////////////////////////////

HANDLE_ERROR(hipMemcpy(input1, &a[n/2], n/4 * sizeof(int), hipMemcpyHostToDevice));

 space = 1;
 direction=0;
 steps = (int)log2((float)n/4);
for (int step = 0; step<steps; step++){
 direction = ((step % 2) !=0);  
Kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(input1, output1, result1, space,  step, steps,direction);
space = space * 2;
}
	Kernel1<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(result1,carry);

	HANDLE_ERROR(hipMemcpy(&c[n/2], result1, n/4 * sizeof(int), hipMemcpyDeviceToHost));

//for(int j =0; j< n/4 ;j++)
//	c[j+n/2]+=carry;

carry = c[(3*n/4)-1]+a[(3*n/4)-1];
//////////////////////////////////////////////////

HANDLE_ERROR(hipMemcpy(input1, &a[3*n/4], n/4 * sizeof(int), hipMemcpyHostToDevice));

 space = 1;
 direction=0;
 steps = (int)log2((float)n/4);
for (int step = 0; step<steps; step++){
 direction = ((step % 2) !=0);  
Kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(input1, output1, result1, space,  step, steps,direction);
space = space * 2;
}
	Kernel1<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(result1,carry);

	HANDLE_ERROR(hipMemcpy(&c[3*n/4], result1, n/4 * sizeof(int), hipMemcpyDeviceToHost));

//for(int j =0; j< n/4 ;j++)
	//c[j+3*n/4]+=carry;



        HANDLE_ERROR(hipFree(input1));
        HANDLE_ERROR(hipFree(output1));
        HANDLE_ERROR(hipFree(result1));

}


if(n==67108864*8)
{

	HANDLE_ERROR(hipMalloc((void**)&input1,  (n/8) * sizeof(int)));
        HANDLE_ERROR(hipMalloc((void**)&output1, (n/8) * sizeof(int)));
        HANDLE_ERROR(hipMalloc((void**)&result1, (n/8) * sizeof(int)));
	
        HANDLE_ERROR(hipMemcpy(input1, a, (n/8) * sizeof(int), hipMemcpyHostToDevice));

	
dim3 THREADS_PER_BLOCK(1024/8,1,1);
	dim3 BLOCKS_PER_GRID( n / (1048576*8), 64,64*2);	

int space = 1;

bool direction=0;
	int steps = (int)log2((float)(n/8));
for ( int step = 0; step<steps; step++){
 direction = ((step % 2) !=0);  
Kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(input1, output1, result1, space,  step, steps,direction);
space = space * 2;
}

	HANDLE_ERROR(hipMemcpy(c, result1, (n/8) * sizeof(int), hipMemcpyDeviceToHost));

int carry = c[(n/8)-1]+a[(n/8)-1];
//////////////////////////////////////////////////

HANDLE_ERROR(hipMemcpy(input1, &a[n/8], n/8 * sizeof(int), hipMemcpyHostToDevice));

 space = 1;
 direction=0;
 steps = (int)log2((float)n/8);
for (int step = 0; step<steps; step++){
 direction = ((step % 2) !=0);  
Kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(input1, output1, result1, space,  step, steps,direction);
space = space * 2;
}
	Kernel1<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(result1,carry);
	HANDLE_ERROR(hipMemcpy(&c[n/8], result1, n/8 * sizeof(int), hipMemcpyDeviceToHost));

//for(int j =0; j< n/8 ;j++)
//	c[j+n/8]+=carry;

 carry = c[(n/4)-1]+a[(n/4)-1];
//////////////////////////////////////////////////

HANDLE_ERROR(hipMemcpy(input1, &a[n/4], n/8 * sizeof(int), hipMemcpyHostToDevice));

 space = 1;
 direction=0;
 steps = (int)log2((float)n/8);
for (int step = 0; step<steps; step++){
 direction = ((step % 2) !=0);  
Kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(input1, output1, result1, space,  step, steps,direction);
space = space * 2;
}
	Kernel1<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(result1,carry);
	HANDLE_ERROR(hipMemcpy(&c[n/4], result1, n/8 * sizeof(int), hipMemcpyDeviceToHost));

//for(int j =0; j< n/8 ;j++)
//	c[j+n/4]+=carry;

carry = c[(3*n/8)-1]+a[(3*n/8)-1];
//////////////////////////////////////////////////

HANDLE_ERROR(hipMemcpy(input1, &a[3*n/8], n/8 * sizeof(int), hipMemcpyHostToDevice));

 space = 1;
 direction=0;
 steps = (int)log2((float)n/8);
for (int step = 0; step<steps; step++){
 direction = ((step % 2) !=0);  
Kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(input1, output1, result1, space,  step, steps,direction);
space = space * 2;
}
	Kernel1<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(result1,carry);

	HANDLE_ERROR(hipMemcpy(&c[3*n/8], result1, n/8 * sizeof(int), hipMemcpyDeviceToHost));

//for(int j =0; j< n/8 ;j++)
//	c[j+3*n/8]+=carry;

carry = c[(n/8)*4-1]+a[(n/8)*4-1];
//////////////////////////////////////////////////

HANDLE_ERROR(hipMemcpy(input1, &a[(n/8)*4], n/8 * sizeof(int), hipMemcpyHostToDevice));

 space = 1;
 direction=0;
 steps = (int)log2((float)n/8);
for (int step = 0; step<steps; step++){
 direction = ((step % 2) !=0);  
Kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(input1, output1, result1, space,  step, steps,direction);
space = space * 2;
}
	Kernel1<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(result1,carry);
	HANDLE_ERROR(hipMemcpy(&c[(n/8)*4], result1, n/8 * sizeof(int), hipMemcpyDeviceToHost));

//for(int j =0; j< n/8 ;j++)
//	c[j+(n/8)*4]+=carry;

carry = c[(n/8)*5-1]+a[(n/8)*5-1];
//////////////////////////////////////////////////

HANDLE_ERROR(hipMemcpy(input1, &a[(n/8)*5], n/8 * sizeof(int), hipMemcpyHostToDevice));

 space = 1;
 direction=0;
 steps = (int)log2((float)n/8);
for (int step = 0; step<steps; step++){
 direction = ((step % 2) !=0);  
Kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(input1, output1, result1, space,  step, steps,direction);
space = space * 2;
}
	Kernel1<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(result1,carry);
	HANDLE_ERROR(hipMemcpy(&c[(n/8)*5], result1, n/8 * sizeof(int), hipMemcpyDeviceToHost));

//for(int j =0; j< n/8 ;j++)
//	c[j+(n/8)*5]+=carry;

carry = c[(n/8)*6-1]+a[(n/8)*6-1];
//////////////////////////////////////////////////

HANDLE_ERROR(hipMemcpy(input1, &a[(n/8)*6], n/8 * sizeof(int), hipMemcpyHostToDevice));

 space = 1;
 direction=0;
 steps = (int)log2((float)n/8);
for (int step = 0; step<steps; step++){
 direction = ((step % 2) !=0);  
Kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(input1, output1, result1, space,  step, steps,direction);
space = space * 2;
}
	Kernel1<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(result1,carry);
	HANDLE_ERROR(hipMemcpy(&c[(n/8)*6], result1, n/8 * sizeof(int), hipMemcpyDeviceToHost));

//for(int j =0; j< n/8 ;j++)
//	c[j+(n/8)*6]+=carry;

carry = c[(n/8)*7-1]+a[(n/8)*7-1];
//////////////////////////////////////////////////

HANDLE_ERROR(hipMemcpy(input1, &a[(n/8)*7], n/8 * sizeof(int), hipMemcpyHostToDevice));

 space = 1;
 direction=0;
 steps = (int)log2((float)n/8);
for (int step = 0; step<steps; step++){
 direction = ((step % 2) !=0);  
Kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(input1, output1, result1, space,  step, steps,direction);
space = space * 2;
}
	Kernel1<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK >>>(result1,carry);
	HANDLE_ERROR(hipMemcpy(&c[(n/8)*7], result1, n/8 * sizeof(int), hipMemcpyDeviceToHost));

//for(int j =0; j< n/8 ;j++)
//	c[j+(n/8)*7]+=carry;



        HANDLE_ERROR(hipFree(input1));
        HANDLE_ERROR(hipFree(output1));
        HANDLE_ERROR(hipFree(result1));

}







}

