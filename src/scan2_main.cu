#include "hip/hip_runtime.h"
//Do NOT MODIFY THIS FILE
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

#include "gpuerrors.h"
#include "scan2.h"

//-----------------------------------------------------------------------------
void fill(int* data, int size) {			// to fill array with -2 -1 0 1 2
    for (int i=0; i<size; ++i)
        data[i] = (int) (rand() % 3);
}

double calc_mse (int* data1, int* data2, int size) {	// calculate error by (  )^2
	double mse = 0.0;
	int i; 
	int e = 0;
	for (i=0; i<size; i++) {
		e = data1[i] - data2[i];
		e = e * e;
		mse += (double) e;
	}
	mse = mse / ((double)size);
	return mse;
}
//-----------------------------------------------------------------------------
void cpuKernel (int* a, int* c, int n) {		// calculate scan algorithm  
	int i = 0;
	int sum = 0;
	for (i = 0; i < n; i += 1){
		c[i] = sum;			// to have exclusive scan
		sum += (a[i]);
		//c[i] = sum;			// to have inclusive scan
	}
	return;
}
//-----------------------------------------------------------------------------
int main ( int argc, char** argv) {
	   
	int* a;
	int* c_serial;
	int* c;	
	
	int m = 5; 
	int n = 32;

	if (argc > 1){
		m = atoi(argv[1]);
		n = (1 << m);
	}

	a        = (int*)malloc(n * sizeof(int));
	c_serial = (int*)malloc(n * sizeof(int));
	c        = (int*)malloc(n * sizeof(int));
				
	srand(0);
	fill(a, n);

	clock_t t0 = clock(); 
	cpuKernel (a, c_serial, n);
	clock_t t1 = clock(); 
		
	clock_t t2 = clock(); 
	gpuKernel (a, c, n);
	clock_t t3 = clock();
		
	float mse;
	mse = calc_mse( c_serial, c, n );
	
	printf("n=%d\t CPU=%06ld ms GPU=%06ld ms mse=%f\n",n, (t1-t0)/1000, (t3-t2)/1000, mse);	
	//printf("%d\t%d\t%d\t%d\t%d\t", c[0],c[1],c[2],c[3],c[4]);

	free(a);
	free(c_serial);
	free(c);
	return 0;
}
//Do NOT MODIFY THIS FILE

